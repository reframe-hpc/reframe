#include  <stdio.h>
#include "hip/hip_runtime.h"

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

extern "C" {

__global__  void simple_add(float* a, float* b, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x; 
  if(i < n) {
    a[i] = a[i] + b[i];
  }
}

void cuda_kernel_no_copy(float* a, float* b, int n)
{
  const int THREADS_PER_BLOCK = 1;
  const int NUMBER_OF_BLOCKS = 10;

  hipDeviceSynchronize();
  simple_add<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(a, b, n);
  hipDeviceSynchronize();

  cudaCheckErrors("cuda error");
}

void cuda_kernel_with_copy(float* a, float* b, int n)
{
  const int THREADS_PER_BLOCK = 1;
  const int NUMBER_OF_BLOCKS = 10;

  float* d_a;
  float* d_b;
  hipMalloc(&d_a, n*sizeof(float));
  hipMalloc(&d_b, n*sizeof(float));
  hipMemcpy(d_a, a, n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, n*sizeof(float), hipMemcpyHostToDevice);
    
  hipDeviceSynchronize();
  simple_add<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(d_a, d_b, n);
  hipDeviceSynchronize();

  hipMemcpy(a, d_a, n*sizeof(float), hipMemcpyDeviceToHost);
  
  cudaCheckErrors("cuda error");

}
};
