#include "hip/hip_runtime.h"
/*
 * Modifications for CSCS by Mark Klein (klein@cscs.ch)
 * - NVML bindings
 * - Reduced output
 *
 * original gpu_burn
 * Copyright (c) 2016, Ville Timonen
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 *    list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
 * ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * The views and conclusions contained in the software and documentation are those
 * of the authors and should not be interpreted as representing official policies,
 * either expressed or implied, of the FreeBSD Project.
 */

#define SIZE 2048ul // Matrices are SIZE*SIZE..  2048^2 should be efficiently implemented in CUBLAS
#define USEMEM 0.9 // Try to allocate 90% of memory

// Used to report op/s, measured through Visual Profiler, CUBLAS from CUDA 7.5
// (Seems that they indeed take the naive dim^3 approach)
#define OPS_PER_MUL 17188257792ul

#include <cstdio>
#include <string>
#include <map>
#include <vector>
#include <sys/types.h>
#include <signal.h>
#include <sys/wait.h>
#include <string.h>
#include <unistd.h>
#include <time.h>
#include <cstdlib>

#include <nvml.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

// Actually, there are no rounding errors due to results being accumulated in an arbitrary order..
// Therefore EPSILON = 0.0f is OK
#define EPSILON 0.001f
#define EPSILOND 0.0000001

extern "C" __global__ void compareF(float *C, int *faultyElems, size_t iters) {
    size_t iterStep = blockDim.x*blockDim.y*gridDim.x*gridDim.y;
    size_t myIndex = (blockIdx.y*blockDim.y + threadIdx.y)* // Y
		     gridDim.x*blockDim.x + // W
		     blockIdx.x*blockDim.x + threadIdx.x; // X

    int myFaulty = 0;
    for (size_t i = 1; i < iters; ++i)
        if (fabsf(C[myIndex] - C[myIndex + i*iterStep]) > EPSILON)
            myFaulty++;

    atomicAdd(faultyElems, myFaulty);
}

extern "C" __global__ void compareD(double *C, int *faultyElems, size_t iters) {
    size_t iterStep = blockDim.x*blockDim.y*gridDim.x*gridDim.y;
    size_t myIndex = (blockIdx.y*blockDim.y + threadIdx.y)* // Y
                     gridDim.x*blockDim.x + // W
		     blockIdx.x*blockDim.x + threadIdx.x; // X

    int myFaulty = 0;
    for (size_t i = 1; i < iters; ++i)
        if (fabs(C[myIndex] - C[myIndex + i*iterStep]) > EPSILOND)
            myFaulty++;

    atomicAdd(faultyElems, myFaulty);
}


void checkError(int rCode, std::string desc = "") {
    static std::map<int, std::string> g_errorStrings;
    if (!g_errorStrings.size()) {
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorInvalidValue, "hipErrorInvalidValue"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorOutOfMemory, "hipErrorOutOfMemory"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorNotInitialized, "hipErrorNotInitialized"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorDeinitialized, "hipErrorDeinitialized"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorNoDevice, "hipErrorNoDevice"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorInvalidDevice, "hipErrorInvalidDevice"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorInvalidImage, "hipErrorInvalidImage"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorInvalidContext, "hipErrorInvalidContext"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorMapFailed, "hipErrorMapFailed"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorUnmapFailed, "hipErrorUnmapFailed"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorArrayIsMapped, "hipErrorArrayIsMapped"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorAlreadyMapped, "hipErrorAlreadyMapped"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorNoBinaryForGpu, "hipErrorNoBinaryForGpu"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorAlreadyAcquired, "hipErrorAlreadyAcquired"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorNotMapped, "hipErrorNotMapped"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorNotMappedAsArray, "hipErrorNotMappedAsArray"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorNotMappedAsPointer, "hipErrorNotMappedAsPointer"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorUnsupportedLimit, "hipErrorUnsupportedLimit"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorContextAlreadyInUse, "hipErrorContextAlreadyInUse"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorInvalidSource, "hipErrorInvalidSource"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorFileNotFound, "hipErrorFileNotFound"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorSharedObjectSymbolNotFound, "hipErrorSharedObjectSymbolNotFound"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorSharedObjectInitFailed, "hipErrorSharedObjectInitFailed"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorOperatingSystem, "hipErrorOperatingSystem"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorInvalidHandle, "hipErrorInvalidHandle"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorNotFound, "hipErrorNotFound"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorNotReady, "hipErrorNotReady"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorLaunchFailure, "hipErrorLaunchFailure"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorLaunchOutOfResources, "hipErrorLaunchOutOfResources"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorLaunchTimeOut, "hipErrorLaunchTimeOut"));
        g_errorStrings.insert(std::pair<int, std::string>(CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING, "CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorSetOnActiveProcess, "hipErrorSetOnActiveProcess"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorContextIsDestroyed, "hipErrorContextIsDestroyed"));
        g_errorStrings.insert(std::pair<int, std::string>(hipErrorUnknown, "hipErrorUnknown"));
    }

    if (rCode != hipSuccess)
        throw ((desc == "") ?
            std::string("Error: ") :
            (std::string("Error in \"") + desc + std::string("\": "))) + g_errorStrings[rCode];
}

void checkError(hipblasStatus_t rCode, std::string desc = "") {
    static std::map<hipblasStatus_t, std::string> g_errorStrings;
    if (!g_errorStrings.size()) {
        g_errorStrings.insert(std::pair<hipblasStatus_t, std::string>(HIPBLAS_STATUS_NOT_INITIALIZED, "HIPBLAS_STATUS_NOT_INITIALIZED"));
        g_errorStrings.insert(std::pair<hipblasStatus_t, std::string>(HIPBLAS_STATUS_ALLOC_FAILED, "HIPBLAS_STATUS_ALLOC_FAILED"));
        g_errorStrings.insert(std::pair<hipblasStatus_t, std::string>(HIPBLAS_STATUS_INVALID_VALUE, "HIPBLAS_STATUS_INVALID_VALUE"));
        g_errorStrings.insert(std::pair<hipblasStatus_t, std::string>(HIPBLAS_STATUS_ARCH_MISMATCH, "HIPBLAS_STATUS_ARCH_MISMATCH"));
        g_errorStrings.insert(std::pair<hipblasStatus_t, std::string>(HIPBLAS_STATUS_MAPPING_ERROR, "HIPBLAS_STATUS_MAPPING_ERROR"));
        g_errorStrings.insert(std::pair<hipblasStatus_t, std::string>(HIPBLAS_STATUS_EXECUTION_FAILED, "HIPBLAS_STATUS_EXECUTION_FAILED"));
        g_errorStrings.insert(std::pair<hipblasStatus_t, std::string>(HIPBLAS_STATUS_INTERNAL_ERROR, "HIPBLAS_STATUS_INTERNAL_ERROR"));
    }

    if (rCode != HIPBLAS_STATUS_SUCCESS)
        throw ((desc == "") ?
            std::string("Error: ") :
            (std::string("Error in \"") + desc + std::string("\": "))) + g_errorStrings[rCode];
}

template <class T> class GPU_Test {
    public:
        GPU_Test(int dev, bool doubles) : d_devNumber(dev), d_doubles(doubles) {
            checkError(hipDeviceGet(&d_dev, d_devNumber));
            checkError(hipCtxCreate(&d_ctx, 0, d_dev));
            bind();
            //checkError(cublasInit());
            checkError(hipblasCreate(&d_cublas), "init");
            d_error = 0;
        }
	~GPU_Test() {
            bind();
            checkError(hipFree(d_Cdata), "Free A");
            checkError(hipFree(d_Adata), "Free B");
            checkError(hipFree(d_Bdata), "Free C");
            // printf("Freed memory for dev %d\n", d_devNumber);
            hipblasDestroy(d_cublas);
            // printf("Uninitted cublas\n");
        }

        unsigned long long int getErrors() {
            unsigned long long int tempErrs = d_error;
            d_error = 0;
            return tempErrs;
        }

        size_t getIters() {
            return d_iters;
        }

        void bind() {
            checkError(hipCtxSetCurrent(d_ctx), "Bind CTX");
        }

        size_t totalMemory() {
            bind();
            size_t freeMem, totalMem;
            checkError(hipMemGetInfo(&freeMem, &totalMem));
            return totalMem;
        }

        size_t availMemory() {
            bind();
            size_t freeMem, totalMem;
            checkError(hipMemGetInfo(&freeMem, &totalMem));
            return freeMem;
        }

        void initBuffers(T *A, T *B) {
            bind();
            size_t useBytes = (size_t)((double)availMemory()*USEMEM);
            // printf("Initialized device %d with %lu MB of memory (%lu MB available, using %lu MB of it), %s\n",
            // d_devNumber, totalMemory()/1024ul/1024ul, availMemory()/1024ul/1024ul, useBytes/1024ul/1024ul,
            // d_doubles ? "using DOUBLES" : "using FLOATS");
            size_t d_resultSize = sizeof(T)*SIZE*SIZE;
            d_iters = (useBytes - 2*d_resultSize)/d_resultSize; // We remove A and B sizes
            // printf("Results are %d bytes each, thus performing %d iterations\n", d_resultSize, d_iters);
            checkError(hipMalloc(&d_Cdata, d_iters*d_resultSize), "C alloc");
            checkError(hipMalloc(&d_Adata, d_resultSize), "A alloc");
            checkError(hipMalloc(&d_Bdata, d_resultSize), "B alloc");
            checkError(hipMalloc(&d_faultyElemData, sizeof(int)), "faulty data");
            // Populating matrices A and B
            checkError(hipMemcpyHtoD(d_Adata, A, d_resultSize), "A -> device");
            checkError(hipMemcpyHtoD(d_Bdata, B, d_resultSize), "A -> device");
            // initCompareKernel();
        }

	void compute() {
            bind();
            static const float alpha = 1.0f;
            static const float beta = 0.0f;
            static const double alphaD = 1.0;
            static const double betaD = 0.0;

            for (size_t i = 0; i < d_iters; ++i) {
                if (d_doubles)
                    checkError(hipblasDgemm(d_cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                           SIZE, SIZE, SIZE, &alphaD,
                                           (const double*)d_Adata, SIZE,
                                           (const double*)d_Bdata, SIZE,
                                           &betaD,
                                           (double*)d_Cdata + i*SIZE*SIZE, SIZE), "DGEMM");
                else
                    checkError(hipblasSgemm(d_cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                           SIZE, SIZE, SIZE, &alpha,
                                           (const float*)d_Adata, SIZE,
                                           (const float*)d_Bdata, SIZE,
                                           &beta,
                                           (float*)d_Cdata + i*SIZE*SIZE, SIZE), "SGEMM");
            }
        }

        void compare() {
            int faultyElems;
            checkError(hipMemsetD32(d_faultyElemData, 0, 1), "memset");
            dim3 block(g_blockSize,g_blockSize);
            dim3 grid(SIZE/g_blockSize,SIZE/g_blockSize);
            //checkError(cuLaunchGrid(d_function, SIZE/g_blockSize, SIZE/g_blockSize), "Launch grid");
            if(d_doubles)
                compareD<<<grid,block>>>((double*)d_Cdata,(int*)d_faultyElemData,(size_t)d_iters);
            else
                compareF<<<grid,block>>>((float*)d_Cdata,(int*)d_faultyElemData,(size_t)d_iters);

            checkError(hipMemcpyDtoH(&faultyElems, d_faultyElemData, sizeof(int)), "Read faultyelemdata");
            if (faultyElems) {
                d_error += (long long int)faultyElems;
                printf("WE FOUND %d FAULTY ELEMENTS from GPU %d\n", faultyElems, d_devNumber);
            }
        }

        private:
            bool d_doubles;
            int d_devNumber;
            size_t d_iters;
            size_t d_resultSize;

            long long int d_error;

            static const int g_blockSize = 16;

            hipDevice_t d_dev;
            hipCtx_t d_ctx;
            hipModule_t d_module;
            hipFunction_t d_function;

            hipDeviceptr_t d_Cdata;
            hipDeviceptr_t d_Adata;
            hipDeviceptr_t d_Bdata;
            hipDeviceptr_t d_faultyElemData;

            hipblasHandle_t d_cublas;
};

// Returns the number of devices
int initCuda() {
    checkError(hipInit(0));
    int deviceCount = 0;
    checkError(hipGetDeviceCount(&deviceCount));

    if (!deviceCount)
        throw std::string("No CUDA devices");

        #ifdef USEDEV
        if (USEDEV >= deviceCount)
            throw std::string("Not enough devices for USEDEV");
        #endif

    return deviceCount;
}

template<class T> void startBurn(int index, int writeFd, T *A, T *B, bool doubles) {
    GPU_Test<T> *our;
    try {
        our = new GPU_Test<T>(index, doubles);
        our->initBuffers(A, B);
    }
    catch (std::string e) {
        fprintf(stderr, "Couldn't init a GPU test: %s\n", e.c_str());
        exit(124);
    }

    // The actual work
    /*int iters = 0;
    unsigned long long int errors = 0;*/
    try {
        while (true) {
            our->compute();
            our->compare();
            /*errors += our->getErrors();
            iters++;*/
            int ops = our->getIters();
            write(writeFd, &ops, sizeof(int));
            ops = our->getErrors();
            write(writeFd, &ops, sizeof(int));
        }
    }
    catch (std::string e) {
    fprintf(stderr, "Failure during compute: %s\n", e.c_str());
    int ops = -1;
    // Signalling that we failed
    write(writeFd, &ops, sizeof(int));
    write(writeFd, &ops, sizeof(int));
    exit(111);
    }
}

void updateTemps(std::vector<int> *temps) {
    const int readSize = 10240;
    static int gpuIter = 0;
    char data[readSize+1];
    unsigned int device_count, i;
    int curPos = 0;
    nvmlInit();
    nvmlDeviceGetCount(&device_count);
    for (i = 0; i < device_count; i++) {
        nvmlDevice_t device;
        nvmlDeviceGetHandleByIndex(i, &device);
        unsigned int tempValue;
        nvmlDeviceGetTemperature ( device, NVML_TEMPERATURE_GPU, &tempValue);
        temps->at(gpuIter) = (int)tempValue;
        gpuIter = (gpuIter+1)%(temps->size());
    }
}

void listenClients(std::vector<int> clientFd, std::vector<pid_t> clientPid, int runTime) {
    fd_set waitHandles;

    // pid_t tempPid;
    char hostname[256];
    hostname[255]='\0';
    gethostname(hostname,255);
    int maxHandle = 0;
    FD_ZERO(&waitHandles);

    for (size_t i = 0; i < clientFd.size(); ++i) {
        if (clientFd.at(i) > maxHandle)
            maxHandle = clientFd.at(i);
        FD_SET(clientFd.at(i), &waitHandles);
    }

    std::vector<int> clientTemp;
    std::vector<int> clientErrors;
    std::vector<int> clientCalcs;
    std::vector<struct timespec> clientUpdateTime;
    std::vector<float> clientGflops;
    std::vector<bool> clientFaulty;

    time_t startTime = time(0);

    for (size_t i = 0; i < clientFd.size(); ++i) {
        clientTemp.push_back(0);
        clientErrors.push_back(0);
        clientCalcs.push_back(0);
        struct timespec thisTime;
        clock_gettime(CLOCK_REALTIME, &thisTime);
        clientUpdateTime.push_back(thisTime);
        clientGflops.push_back(0.0f);
        clientFaulty.push_back(false);
    }

    int changeCount;
    float nextReport = 2.0f;
    bool childReport = false;
    while ((changeCount = select(maxHandle+1, &waitHandles, NULL, NULL, NULL))) {
        size_t thisTime = time(0);
        struct timespec thisTimeSpec;
        clock_gettime(CLOCK_REALTIME, &thisTimeSpec);

        // printf("got new data! %d\n", changeCount);
        // Going through all descriptors
        for (size_t i = 0; i < clientFd.size(); ++i)
            if (FD_ISSET(clientFd.at(i), &waitHandles)) {
                // First, reading processed
                int processed, errors;
                read(clientFd.at(i), &processed, sizeof(int));
                // Then errors
                read(clientFd.at(i), &errors, sizeof(int));

                clientErrors.at(i) += errors;
                if (processed == -1)
                    clientCalcs.at(i) = -1;
                else {
                    double flops = (double)processed * (double)OPS_PER_MUL;
                    struct timespec clientPrevTime = clientUpdateTime.at(i);
                    double clientTimeDelta = (double)thisTimeSpec.tv_sec + (double)thisTimeSpec.tv_nsec / 1000000000.0 - ((double)clientPrevTime.tv_sec + (double)clientPrevTime.tv_nsec / 1000000000.0);
                    clientUpdateTime.at(i) = thisTimeSpec;

                    clientGflops.at(i) = (double)((unsigned long long int)processed * OPS_PER_MUL) / clientTimeDelta / 1000.0 / 1000.0 / 1000.0;
                    clientCalcs.at(i) += processed;
                }

                childReport = true;
            }

            // if (FD_ISSET(tempHandle, &waitHandles))
            updateTemps(&clientTemp);

            // Resetting the listeners
            FD_ZERO(&waitHandles);
            // FD_SET(tempHandle, &waitHandles);
            for (size_t i = 0; i < clientFd.size(); ++i)
                FD_SET(clientFd.at(i), &waitHandles);

            // Printing progress (if a child has initted already)
            if (childReport) {
                float elapsed = fminf((float)(thisTime-startTime)/(float)runTime*100.0f, 100.0f);
                // printf("\r%s: ",hostname);
                // printf("%.1f%%  ", elapsed);
                // printf("proc'd: ");
                // for (size_t i = 0; i < clientCalcs.size(); ++i) {
                // printf("%d (%.0f Gflop/s) ", clientCalcs.at(i), clientGflops.at(i));
                // if (i != clientCalcs.size() - 1)
                // printf("- ");
                // }
                // printf("  errors: ");
                for (size_t i = 0; i < clientErrors.size(); ++i) {
                    std::string note = "%d ";
                    // if (clientCalcs.at(i) == -1)
                    // note += " (DIED!)";
                    // else if (clientErrors.at(i))
                    // note += " (WARNING!)";

                    // printf(note.c_str(), clientErrors.at(i));
                    // if (i != clientCalcs.size() - 1)
                    // printf("- ");
                }
                // printf("  temps: ");
                // for (size_t i = 0; i < clientTemp.size(); ++i) {
                // printf(clientTemp.at(i) != 0 ? "%d C " : "-- ", clientTemp.at(i));
                // if (i != clientCalcs.size() - 1)
                // printf("- ");
                // }

                fflush(stdout);

                if (nextReport < elapsed) {
                    nextReport = elapsed + 2.0f;
                    // printf("\n\tSummary at:   ");
                    // fflush(stdout);
                    // system("date"); // Printing a date
                    // fflush(stdout);
                    // printf("\n");
                    // printf("\t(checkpoint)\n");
                    for (size_t i = 0; i < clientErrors.size(); ++i) {
                        if (clientErrors.at(i))
                            clientFaulty.at(i) = true;
                        clientErrors.at(i) = 0;
                    }
                }
            }

            // Checking whether all clients are dead
            bool oneAlive = false;
            for (size_t i = 0; i < clientCalcs.size(); ++i)
                if (clientCalcs.at(i) != -1)
                    oneAlive = true;
            if (!oneAlive) {
                fprintf(stderr, "\n\nNo clients are alive!  Aborting\n");
                exit(123);
            }

            if (startTime + runTime < thisTime)
                break;
    }

    // printf("\nKilling processes.. ");
    fflush(stdout);
    for (size_t i = 0; i < clientPid.size(); ++i)
        kill(clientPid.at(i), 15);

    // kill(tempPid, 15);
    // close(tempHandle);

    while (wait(NULL) != -1);
    printf("Node %s:\n", hostname);

    // printf(" Tested %d GPUs: ", (int)clientPid.size());
    for (size_t i = 0; i < clientPid.size(); ++i) {
        printf("  GPU %2d(%s): %4.0f GF/s  %i Celsius\n", (int)i,clientFaulty.at(i) ? "FAULTY" : "OK", clientGflops.at(i), clientTemp.at(i));
    }
    printf("\n");
}

template<class T> void launch(int runLength, bool useDoubles) {
    //    std::system("nvidia-smi -L");

    // Initializing A and B with random data
    T *A = (T*) malloc(sizeof(T)*SIZE*SIZE);
    T *B = (T*) malloc(sizeof(T)*SIZE*SIZE);
    srand(10);
    for (size_t i = 0; i < SIZE*SIZE; ++i) {
        A[i] = (T)((double)(rand()%1000000)/100000.0);
        B[i] = (T)((double)(rand()%1000000)/100000.0);
    }

    // Forking a process..  This one checks the number of devices to use,
    // returns the value, and continues to use the first one.
    int mainPipe[2];
    pipe(mainPipe);
    int readMain = mainPipe[0];
    std::vector<int> clientPipes;
    std::vector<pid_t> clientPids;
    clientPipes.push_back(readMain);

    pid_t myPid = fork();
    if (!myPid) {
        // Child
        close(mainPipe[0]);
        int writeFd = mainPipe[1];
        int devCount = initCuda();
        write(writeFd, &devCount, sizeof(int));
        startBurn<T>(0, writeFd, A, B, useDoubles);
        close(writeFd);
        return;
    }
    else {
        clientPids.push_back(myPid);
        close(mainPipe[1]);
        int devCount;
        read(readMain, &devCount, sizeof(int));

        if (!devCount) {
            fprintf(stderr, "No CUDA devices\n");
        }
        else {
            for (int i = 1; i < devCount; ++i) {
                int slavePipe[2];
                pipe(slavePipe);
                clientPipes.push_back(slavePipe[0]);
                pid_t slavePid = fork();
                if (!slavePid) {
                    // Child
                    close(slavePipe[0]);
                    initCuda();
                    startBurn<T>(i, slavePipe[1], A, B, useDoubles);
                    close(slavePipe[1]);
                    return;
                }
                else {
                    clientPids.push_back(slavePid);
                    close(slavePipe[1]);
                }
            }
            listenClients(clientPipes, clientPids, runLength);
        }
    }

    for (size_t i = 0; i < clientPipes.size(); ++i)
        close(clientPipes.at(i));

    free(A);
    free(B);
}

int main(int argc, char **argv) {
    int runLength = 10;
    bool useDoubles = false;
    int thisParam = 0;
    if (argc >= 2 && std::string(argv[1]) == "-d") {
        useDoubles = true;
        thisParam++;
    }
    if (argc-thisParam < 2)
        printf("Run length not specified in the command line.  Burning for 10 secs\n");
    else
        runLength = atoi(argv[1+thisParam]);

    if (useDoubles)
        launch<double>(runLength, useDoubles);
    else
        launch<float>(runLength, useDoubles);

    return 0;
}
